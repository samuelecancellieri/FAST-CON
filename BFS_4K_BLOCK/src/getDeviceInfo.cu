
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
    std::cout << devProp.major * 10 + devProp.minor
              << ";" << devProp.multiProcessorCount;
}
