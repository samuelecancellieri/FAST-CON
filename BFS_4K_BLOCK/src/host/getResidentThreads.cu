#include <hip/hip_runtime.h>
#include <iostream>


int main(int argc, char* argv[]) {
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	if (argv[1][0] == '0')
		std::cout << devProp.multiProcessorCount * devProp.maxThreadsPerMultiProcessor;
}
